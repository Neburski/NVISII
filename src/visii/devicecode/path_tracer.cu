#include "hip/hip_runtime.h"
#include "path_tracer.h"
#include "disney_bsdf.h"
#include "lights.h"
#include "launch_params.h"
#include "types.h"
#include <optix_device.h>
#include <owl/common/math/random.h>

typedef owl::common::LCG<4> Random;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct RayPayload {
    uint32_t entityID;
    float2 uv;
    float tHit;
    float3 normal;
    float3 gnormal;
    // float pad;
};

inline __device__
float3 missColor(const owl::Ray &ray)
{
  auto pixelID = owl::getLaunchIndex();

  float3 rayDir = normalize(ray.direction);
  float t = 0.5f*(rayDir.z + 1.0f);
  float3 c = (1.0f - t) * make_float3(1.0f, 1.0f, 1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
    RayPayload &payload = get_payload<RayPayload>();
    payload.tHit = -1.f;
    payload.entityID = -1;
    owl::Ray ray;
    ray.direction = optixGetWorldRayDirection();
    payload.normal = missColor(ray) * optixLaunchParams.domeLightIntensity;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    
    const float2 bc    = optixGetTriangleBarycentrics();
    const int instID   = optixGetInstanceIndex();
    const int primID   = optixGetPrimitiveIndex();
    const int entityID = optixLaunchParams.instanceToEntityMap[instID];
    const ivec3 index  = self.index[primID];
    
    // compute position: (actually not needed. implicit via tMax )
    // vec3 V;
    // {
    //     const vec3 &A      = self.vertex[index.x];
    //     const vec3 &B      = self.vertex[index.y];
    //     const vec3 &C      = self.vertex[index.z];
    //     V = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    // }

    // compute normal:
    float3 N, GN;

    const float3 &A      = (float3&) self.vertex[index.x];
    const float3 &B      = (float3&) self.vertex[index.y];
    const float3 &C      = (float3&) self.vertex[index.z];
    GN = normalize(cross(B-A,C-A));
    
    if (self.normals) {
        const float3 &A = (float3&) self.normals[index.x];
        const float3 &B = (float3&) self.normals[index.y];
        const float3 &C = (float3&) self.normals[index.z];
        N = normalize(A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y);
    } else {
        N = GN;
    }

    GN = normalize(optixTransformNormalFromObjectToWorldSpace(GN));
    N = normalize(optixTransformNormalFromObjectToWorldSpace(N));
    // normalize(transpose(mat3(gl_WorldToObjectNV)) * payload.m_n);
    // N  = normalize(transpose(mat3(gl_WorldToObjectNV)) * payload.m_n);

    // compute uv:
    float2 UV;
    if (self.texcoords) {
        const float2 &A = (float2&) self.texcoords[index.x];
        const float2 &B = (float2&) self.texcoords[index.y];
        const float2 &C = (float2&) self.texcoords[index.z];
        UV = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    } else {
        UV = bc;
    }

    // store data in payload
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.entityID = entityID;
    prd.uv = UV;
    prd.tHit = optixGetRayTmax();
    prd.normal = N;
    prd.gnormal = GN;
}

inline __device__
bool loadCamera(EntityStruct &cameraEntity, CameraStruct &camera, TransformStruct &transform)
{
    cameraEntity = optixLaunchParams.cameraEntity;
    if (!cameraEntity.initialized) return false;
    if ((cameraEntity.transform_id < 0) || (cameraEntity.transform_id >= MAX_TRANSFORMS)) return false;
    if ((cameraEntity.camera_id < 0) || (cameraEntity.camera_id >= MAX_CAMERAS)) return false;
    camera = optixLaunchParams.cameras[cameraEntity.camera_id];
    transform = optixLaunchParams.transforms[cameraEntity.transform_id];
    return true;
}

__device__ 
void loadMaterial(const MaterialStruct &p, float2 uv, DisneyMaterial &mat) {

    // uint32_t mask = __float_as_int(p.base_color.x);
    // if (IS_TEXTURED_PARAM(mask)) {
    //     const uint32_t tex_id = GET_TEXTURE_ID(mask);
    //     mat.base_color = make_float3(tex2D<float4>(launch_params.textures[tex_id], uv.x, uv.y));
    // } else {
        mat.base_color = make_float3(p.base_color.x, p.base_color.y, p.base_color.z);
    // }

    mat.metallic = /*textured_scalar_param(*/p.metallic/*, uv)*/;
    mat.specular = /*textured_scalar_param(*/p.specular/*, uv)*/;
    mat.roughness = /*textured_scalar_param(*/p.roughness/*, uv)*/;
    mat.specular_tint = /*textured_scalar_param(*/p.specular_tint/*, uv)*/;
    mat.anisotropy = /*textured_scalar_param(*/p.anisotropic/*, uv)*/;
    mat.sheen = /*textured_scalar_param(*/p.sheen/*, uv)*/;
    mat.sheen_tint = /*textured_scalar_param(*/p.sheen_tint/*, uv)*/;
    mat.clearcoat = /*textured_scalar_param(*/p.clearcoat/*, uv)*/;
    mat.clearcoat_gloss = /*textured_scalar_param(*/1.0 - p.clearcoat_roughness/*, uv)*/;
    mat.ior = /*textured_scalar_param(*/p.ior/*, uv)*/;
    mat.specular_transmission = /*textured_scalar_param(*/p.transmission/*, uv)*/;
    mat.flatness = p.subsurface;
}

inline __device__
owl::Ray generateRay(const CameraStruct &camera, const TransformStruct &transform, ivec2 pixelID, ivec2 frameSize, LCGRand &rng)
{
    /* Generate camera rays */    
    mat4 camWorldToLocal = transform.worldToLocal;
    mat4 projinv = camera.projinv;//glm::inverse(glm::perspective(.785398, 1.0, .1, 1000));//camera.projinv;
    mat4 viewinv = camera.viewinv * camWorldToLocal;
    vec2 aa = vec2(lcg_randomf(rng),lcg_randomf(rng)) - vec2(.5f,.5f);
    vec2 inUV = (vec2(pixelID.x, pixelID.y) + aa) / vec2(optixLaunchParams.frameSize);
    vec3 right = normalize(glm::vec3(viewinv[0]));
    vec3 up = normalize(glm::vec3(viewinv[1]));
    
    float cameraLensRadius = camera.apertureDiameter;

    vec3 p(0.f);
    if (cameraLensRadius > 0.0) {
        do {
            p = 2.0f*vec3(lcg_randomf(rng),lcg_randomf(rng),0.f) - vec3(1.f,1.f,0.f);
        } while (dot(p,p) >= 1.0f);
    }

    vec3 rd = cameraLensRadius * p;
    vec3 lens_offset = (right * rd.x) / float(frameSize.x) + (up * rd.y) / float(frameSize.y);

    vec3 origin = vec3(viewinv * vec4(0.f,0.f,0.f,1.f)) + lens_offset;
    vec2 dir = inUV * 2.f - 1.f; dir.y *= -1.f;
    vec4 t = (projinv * vec4(dir.x, dir.y, -1.f, 1.f));
    vec3 target = vec3(t) / float(t.w);
    vec3 direction = normalize(vec3(viewinv * vec4(target, 0.f))) * camera.focalDistance;
    direction = normalize(direction - lens_offset);

    owl::Ray ray;
    ray.tmin = .001f;
    ray.tmax = 1e20f;//10000.0f;
    ray.origin = owl::vec3f(origin.x, origin.y, origin.z) ;
    ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    ray.direction = normalize(owl::vec3f(direction.x, direction.y, direction.z));
    
    return ray;
}

__device__ float3 sample_direct_light(const DisneyMaterial &mat, const float3 &hit_p,
    const float3 &n, const float3 &v_x, const float3 &v_y, const float3 &w_o,
    const LightStruct *lights, const EntityStruct *entities, const TransformStruct *transforms,
    const uint32_t* light_entities, const uint32_t num_lights, 
    uint16_t &ray_count, LCGRand &rng)
{
    float3 illum = make_float3(0.f);
    
    if (num_lights == 0) return illum;

    uint32_t random_id = lcg_randomf(rng) * num_lights;
    random_id = min(random_id, num_lights - 1);
    uint32_t light_entity_id = light_entities[random_id];
    EntityStruct light_entity = entities[light_entity_id];
    
    // shouldn't happen, but just in case...
    if ((light_entity.light_id < 0) || (light_entity.light_id > MAX_LIGHTS)) return illum;
    if ((light_entity.transform_id < 0) || (light_entity.transform_id > MAX_LIGHTS)) return illum;
    
    LightStruct light = lights[light_entity.light_id];
    TransformStruct transform = transforms[light_entity.transform_id];
    float3 light_emission = make_float3(light.r, light.g, light.b) * light.intensity;

    const uint32_t occlusion_flags = OPTIX_RAY_FLAG_DISABLE_ANYHIT
        | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
        | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT;

    // Sample the light to compute an incident light ray to this point
    {
        float3 light_pos = make_float3(
            transform.localToWorld[3][0], 
            transform.localToWorld[3][1], 
            transform.localToWorld[3][2]);
                // sample_quad_light_position(light,
                // make_float2(lcg_randomf(rng), lcg_randomf(rng)));
        float3 light_dir = light_pos - hit_p;
        float light_dist = length(light_dir);
        light_dir = normalize(light_dir);

        float light_pdf = 1.f; //quad_light_pdf(light, light_pos, hit_p, light_dir);
        float bsdf_pdf = disney_pdf(mat, n, w_o, light_dir, v_x, v_y);

        // uint32_t shadow_hit = 1;
        RayPayload payload;
        payload.entityID = light_entity_id;
        owl::Ray ray;
        ray.tmin = EPSILON * 10.f;
        ray.tmax = light_dist;
        ray.origin = owl::vec3f(hit_p.x, hit_p.y, hit_p.z) ;
        ray.direction = owl::vec3f(light_dir.x, light_dir.y, light_dir.z);
        owl::traceRay(  /*accel to trace against*/ optixLaunchParams.world,
                        /*the ray to trace*/ ray,
                        /*prd*/ payload,
                        occlusion_flags);
                            
    //     optixTrace(launch_params.scene, hit_p, light_dir, EPSILON, light_dist, 0.f,
    //             0xff, occlusion_flags, PRIMARY_RAY, 1, OCCLUSION_RAY,
    //             shadow_hit);
    // #ifdef REPORT_RAY_STATS
    //     ++ray_count;
    // #endif
        if (light_pdf >= EPSILON && bsdf_pdf >= EPSILON && payload.entityID != light_entity_id) {
            float3 bsdf = disney_brdf(mat, n, w_o, light_dir, v_x, v_y);
            float w = 1.f; // power_heuristic(1.f, light_pdf, 1.f, bsdf_pdf); // NOTE: TEMPORARILY DIABLING POWER HEURISTIC SINCE FOR NOW ONLY DOING POINT LIGHTS
            illum = bsdf * light_emission * fabs(dot(light_dir, n)) * w / light_pdf;
        }
    }

    // Sample the BRDF to compute a light sample as well
    // {
    //     float3 w_i;
    //     float bsdf_pdf;
    //     float3 bsdf = sample_disney_brdf(mat, n, w_o, v_x, v_y, rng, w_i, bsdf_pdf);

    //     float light_dist;
    //     float3 light_pos;
    //     if (!all_zero(bsdf) && bsdf_pdf >= EPSILON && quad_intersect(light, hit_p, w_i, light_dist, light_pos)) {
    //         float light_pdf = quad_light_pdf(light, light_pos, hit_p, w_i);
    //         if (light_pdf >= EPSILON) {
    //             float w = power_heuristic(1.f, bsdf_pdf, 1.f, light_pdf);
    //             uint32_t shadow_hit = 1;
    //             optixTrace(launch_params.scene, hit_p, w_i, EPSILON, light_dist, 0.f,
    //                     0xff, occlusion_flags, PRIMARY_RAY, 1, OCCLUSION_RAY,
    //                     shadow_hit);
    // // #ifdef REPORT_RAY_STATS
    // //             ++ray_count;
    // // #endif
    //             if (!shadow_hit) {
    //                 illum = illum + bsdf * light_emission * fabs(dot(w_i, n)) * w / bsdf_pdf;
    //             }
    //         }
    //     }
    // }
    return illum;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    auto pixelID = ivec2(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    auto fbOfs = pixelID.x+optixLaunchParams.frameSize.x* ((optixLaunchParams.frameSize.y - 1) -  pixelID.y);
    LCGRand rng = get_rng(optixLaunchParams.frameID);

    EntityStruct    camera_entity;
    TransformStruct camera_transform;
    CameraStruct    camera;
    if (!loadCamera(camera_entity, camera, camera_transform)) {
        optixLaunchParams.fbPtr[fbOfs] = vec4(lcg_randomf(rng), lcg_randomf(rng), lcg_randomf(rng), 1.f);
        return;
    }


    float3 accum_illum = make_float3(0.f);
    #define SPP 4
    for (uint32_t rid = 0; rid < SPP; ++rid) {

        owl::Ray ray = generateRay(camera, camera_transform, pixelID, optixLaunchParams.frameSize, rng);

        DisneyMaterial mat;
        int bounce = 0;
        float3 illum = make_float3(0.f);
        float3 path_throughput = make_float3(1.f);
        uint16_t ray_count = 0;

        do {
            RayPayload payload;
            owl::traceRay(  /*accel to trace against*/ optixLaunchParams.world,
                            /*the ray to trace*/ ray,
                            /*prd*/ payload);
            #ifdef REPORT_RAY_STATS
                ++ray_count;
            #endif

            // if ray misses, interpret normal as "miss color" assigned by miss program
            if (payload.tHit <= 0.f) {
                illum = illum + path_throughput * payload.normal;
                break;
            }

            EntityStruct entity = optixLaunchParams.entities[payload.entityID];
            MaterialStruct entityMaterial = optixLaunchParams.materials[entity.material_id];
            TransformStruct entityTransform = optixLaunchParams.transforms[entity.transform_id];
            loadMaterial(entityMaterial, payload.uv, mat);

            const float3 w_o = -ray.direction;
            const float3 hit_p = ray.origin + payload.tHit * ray.direction;
            float3 v_x, v_y;
            float3 v_z = payload.normal;
            float3 v_gz = payload.gnormal;
            if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
                // prevents differences from geometric and shading normal from creating black artifacts
                v_z = reflect(-v_z, v_gz); 
            }
            if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
                v_z = -v_z;
            }
            ortho_basis(v_x, v_y, v_z);

            illum = illum + path_throughput * 
                sample_direct_light(mat, hit_p, v_z, v_x, v_y, w_o,
                    optixLaunchParams.lights, 
                    optixLaunchParams.entities, 
                    optixLaunchParams.transforms, 
                    optixLaunchParams.lightEntities,
                    optixLaunchParams.numLightEntities, 
                    ray_count, rng);

            float3 w_i;
            float pdf;
            float3 bsdf = sample_disney_brdf(mat, v_z, w_o, v_x, v_y, rng, w_i, pdf);
            if (pdf < EPSILON || all_zero(bsdf)) {
                break;
            }
            path_throughput = path_throughput * bsdf / pdf;

            if (path_throughput.x < EPSILON && path_throughput.y < EPSILON && path_throughput.z < EPSILON) {
                break;
            }

            // vec3 offset = payload.normal * .001f;
            ray.origin = hit_p;// + make_float3(offset.x, offset.y, offset.z);
            ray.direction = w_i;

            ++bounce;

            // if (tprd.tHit > 0.f) {
            //     finalColor = vec3(tprd.normal.x, tprd.normal.y, tprd.normal.z);
            // }
        } while (bounce < MAX_PATH_DEPTH);
        accum_illum = accum_illum + illum;
    }
    accum_illum = accum_illum / float(SPP);


    // finalColor = vec3(ray.direction.x, ray.direction.y, ray.direction.z);
    /* Write AOVs */
    float4 &prev_color = (float4&) optixLaunchParams.accumPtr[fbOfs];
    float4 accum_color = make_float4((accum_illum + float(optixLaunchParams.frameID) * make_float3(prev_color)) / float(optixLaunchParams.frameID + 1), 1.0f);
    optixLaunchParams.accumPtr[fbOfs] = vec4(
        accum_color.x, 
        accum_color.y, 
        accum_color.z, 
        accum_color.w
    );
    optixLaunchParams.fbPtr[fbOfs] = vec4(
        linear_to_srgb(accum_color.x),
        linear_to_srgb(accum_color.y),
        linear_to_srgb(accum_color.z),
        1.0f
    );
}

