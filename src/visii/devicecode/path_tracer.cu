#include "hip/hip_runtime.h"
#include "path_tracer.h"
#include "disney_bsdf.h"
#include "lights.h"
#include "launch_params.h"
#include "types.h"
#include <optix_device.h>
#include <owl/common/math/random.h>

typedef owl::common::LCG<4> Random;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct RayPayload {
    vec2 uv;
    float tHit;
    uint32_t entityID;
    vec3 normal;
    // float pad;
};

inline __device__
vec3 missColor(const owl::Ray &ray)
{
  auto pixelID = owl::getLaunchIndex();

  vec3 rayDir = glm::normalize(glm::vec3(ray.direction.x, ray.direction.y, ray.direction.z));
  float t = 0.5f*(rayDir.z + 1.0f);
  vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
    RayPayload &payload = get_payload<RayPayload>();
    payload.tHit = -1.f;
    owl::Ray ray;
    ray.direction = optixGetWorldRayDirection();
    payload.normal = missColor(ray);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    
    const float2 bc    = optixGetTriangleBarycentrics();
    const int instID   = optixGetInstanceIndex();
    const int primID   = optixGetPrimitiveIndex();
    const int entityID = optixLaunchParams.instanceToEntityMap[instID];
    const ivec3 index  = self.index[primID];
    
    // compute position: (actually not needed. implicit via tMax )
    // vec3 V;
    // {
    //     const vec3 &A      = self.vertex[index.x];
    //     const vec3 &B      = self.vertex[index.y];
    //     const vec3 &C      = self.vertex[index.z];
    //     V = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    // }

    // compute normal:
    vec3 N;
    if (self.normals) {
        const vec3 &A = self.normals[index.x];
        const vec3 &B = self.normals[index.y];
        const vec3 &C = self.normals[index.z];
        N = normalize(A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y);
    } else {
        const vec3 &A      = self.vertex[index.x];
        const vec3 &B      = self.vertex[index.y];
        const vec3 &C      = self.vertex[index.z];
        N = normalize(cross(B-A,C-A));
    }

    // compute uv:
    vec2 UV;
    if (self.texcoords) {
        const vec2 A = self.texcoords[index.x];
        const vec2 B = self.texcoords[index.y];
        const vec2 C = self.texcoords[index.z];
        UV = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    } else {
        UV = vec2(bc.x, bc.y);
    }

    // store data in payload
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.entityID = entityID;
    prd.uv = UV;
    prd.tHit = optixGetRayTmax();
    prd.normal = N;
}

inline __device__
bool loadCamera(EntityStruct &cameraEntity, CameraStruct &camera, TransformStruct &transform)
{
    cameraEntity = optixLaunchParams.cameraEntity;
    if (!cameraEntity.initialized) return false;
    if ((cameraEntity.transform_id < 0) || (cameraEntity.transform_id >= MAX_TRANSFORMS)) return false;
    if ((cameraEntity.camera_id < 0) || (cameraEntity.camera_id >= MAX_CAMERAS)) return false;
    camera = optixLaunchParams.cameras[cameraEntity.camera_id];
    transform = optixLaunchParams.transforms[cameraEntity.transform_id];
    return true;
}

__device__ 
void loadMaterial(const MaterialStruct &p, vec2 uv, DisneyMaterial &mat) {

    // uint32_t mask = __float_as_int(p.base_color.x);
    // if (IS_TEXTURED_PARAM(mask)) {
    //     const uint32_t tex_id = GET_TEXTURE_ID(mask);
    //     mat.base_color = make_float3(tex2D<float4>(launch_params.textures[tex_id], uv.x, uv.y));
    // } else {
        mat.base_color = make_float3(p.base_color.x, p.base_color.y, p.base_color.z);
    // }

    mat.metallic = /*textured_scalar_param(*/p.metallic/*, uv)*/;
    mat.specular = /*textured_scalar_param(*/p.specular/*, uv)*/;
    mat.roughness = /*textured_scalar_param(*/p.roughness/*, uv)*/;
    mat.specular_tint = /*textured_scalar_param(*/p.specular_tint/*, uv)*/;
    mat.anisotropy = /*textured_scalar_param(*/p.anisotropic/*, uv)*/;
    mat.sheen = /*textured_scalar_param(*/p.sheen/*, uv)*/;
    mat.sheen_tint = /*textured_scalar_param(*/p.sheen_tint/*, uv)*/;
    mat.clearcoat = /*textured_scalar_param(*/p.clearcoat/*, uv)*/;
    mat.clearcoat_gloss = /*textured_scalar_param(*/1.0 - p.clearcoat_roughness/*, uv)*/;
    mat.ior = /*textured_scalar_param(*/p.ior/*, uv)*/;
    mat.specular_transmission = /*textured_scalar_param(*/p.transmission/*, uv)*/;
}

inline __device__
owl::Ray generateRay(const CameraStruct &camera, const TransformStruct &transform, ivec2 pixelID, ivec2 frameSize)
{
    /* Generate camera rays */    
    mat4 camWorldToLocal = transform.worldToLocal;
    mat4 projinv = camera.projinv;//glm::inverse(glm::perspective(.785398, 1.0, .1, 1000));//camera.projinv;
    mat4 viewinv = camera.viewinv * camWorldToLocal;
    vec2 inUV = vec2(pixelID.x, pixelID.y) / vec2(optixLaunchParams.frameSize);
    // if (optixLaunchParams.zoom > 0.f) {
    //     inUV /= optixLaunchParams.zoom;
    //     inUV += (.5f - (.5f / optixLaunchParams.zoom));
    // }

    vec3 origin = vec3(viewinv * vec4(0.f,0.f,0.f,1.f));

    vec2 dir = inUV * 2.f - 1.f; dir.y *= -1.f;
    vec4 t = (projinv * vec4(dir.x, dir.y, -1.f, 1.f));
    vec3 target = vec3(t) / float(t.w);
    vec3 direction = vec3(viewinv * vec4(target, 0.f));
    direction = normalize(direction);

    owl::Ray ray;
    ray.tmin = .001f;
    ray.tmax = 1e20f;//10000.0f;
    ray.origin = owl::vec3f(origin.x, origin.y, origin.z);
    ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    // ray.direction = owl::vec3f(0.0, 1.0, 0.0); // testing...
    // if ((pixelID.x == 0) && (pixelID.y == 0)) {
    //     // printf("dir: %f %f %f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    //     printf("viewinv: %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f\n", 
    //         viewinv[0][0], viewinv[0][1], viewinv[0][2], viewinv[0][3],
    //         viewinv[1][0], viewinv[1][1], viewinv[1][2], viewinv[1][3],
    //         viewinv[2][0], viewinv[2][1], viewinv[2][2], viewinv[2][3],
    //         viewinv[3][0], viewinv[3][1], viewinv[3][2], viewinv[3][3]
    //     );
    // }

    ray.direction = normalize(owl::vec3f(direction.x, direction.y, direction.z));
    // ray.direction = normalize(owl::vec3f(target.x, target.y, target.z));

    // vec3 lookFrom = origin;//(-4.f,-3.f,-2.f);
    // vec3 lookAt(0.f,0.f,0.f);
    // vec3 lookUp(0.f,0.f,1.f);
    // float cosFovy = 0.66f;
    // vec3 camera_pos = lookFrom;
    // vec3 camera_d00
    //   = normalize(lookAt-lookFrom);
    // float aspect = frameSize.x / float(frameSize.y);
    // vec3 camera_ddu
    //   = cosFovy * aspect * normalize(cross(camera_d00,lookUp));
    // vec3 camera_ddv
    //   = cosFovy * normalize(cross(camera_ddu,camera_d00));
    // camera_d00 -= 0.5f * camera_ddu;
    // camera_d00 -= 0.5f * camera_ddv;

    // direction 
    // = normalize(camera_d00
    //             + inUV.x * camera_ddu
    //             + inUV.y * camera_ddv);
    // ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    return ray;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    auto pixelID = ivec2(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    auto fbOfs = pixelID.x+optixLaunchParams.frameSize.x* ((optixLaunchParams.frameSize.y - 1) -  pixelID.y);
    LCGRand rng = get_rng(optixLaunchParams.frameID);

    EntityStruct    camera_entity;
    TransformStruct camera_transform;
    CameraStruct    camera;
    if (!loadCamera(camera_entity, camera, camera_transform)) {
        optixLaunchParams.fbPtr[fbOfs] = vec4(lcg_randomf(rng), lcg_randomf(rng), lcg_randomf(rng), 1.f);
        return;
    }
    owl::Ray ray = generateRay(camera, camera_transform, pixelID, optixLaunchParams.frameSize);

    DisneyMaterial mat;
    int bounce = 0;
    vec3 illum = vec3(0.f);
    vec3 path_throughput = vec3(1.f);
    uint16_t ray_count = 0;

    do {
        RayPayload payload;
        owl::traceRay(  /*accel to trace against*/ optixLaunchParams.world,
                        /*the ray to trace*/ ray,
                        /*prd*/ payload);
        #ifdef REPORT_RAY_STATS
            ++ray_count;
        #endif

        // if ray misses, interpret normal as "miss color" assigned by miss program
        if (payload.tHit <= 0.f) {
            illum = illum + path_throughput * payload.normal;
            break;
        }

        EntityStruct entity = optixLaunchParams.entities[payload.entityID];
        MaterialStruct entityMaterial = optixLaunchParams.materials[entity.material_id];
        loadMaterial(entityMaterial, payload.uv, mat);

        const float3 w_o = -ray.direction;
        const float3 hit_p = ray.origin + payload.tHit * ray.direction;
        float3 v_x, v_y;
        float3 v_z = make_float3(payload.normal.x,payload.normal.y,payload.normal.z);
        if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
            v_z = -v_z;
        }
        ortho_basis(v_x, v_y, v_z);

        // illum = illum + path_throughput * sample_direct_light(mat, hit_p, v_z, v_x, v_y, w_o,
                // params.lights, params.num_lights, ray_count, rng);

        float3 w_i;
        float pdf;
        float3 bsdf = sample_disney_brdf(mat, v_z, w_o, v_x, v_y, rng, w_i, pdf);
        if (pdf < EPSILON || all_zero(bsdf)) {
            break;
        }
        path_throughput = path_throughput * vec3(bsdf.x, bsdf.y, bsdf.z) * fabs(dot(w_i, v_z)) / pdf;

        if (path_throughput.x < EPSILON && path_throughput.y < EPSILON && path_throughput.z < EPSILON) {
            break;
        }

        // vec3 offset = payload.normal * .001f;
        ray.origin = hit_p;// + make_float3(offset.x, offset.y, offset.z);
        ray.direction = w_i;

        ++bounce;

        // if (tprd.tHit > 0.f) {
        //     finalColor = vec3(tprd.normal.x, tprd.normal.y, tprd.normal.z);
        // }
    } while (bounce < MAX_PATH_DEPTH);

    // finalColor = vec3(ray.direction.x, ray.direction.y, ray.direction.z);
    /* Write AOVs */
    vec4 prev_color = optixLaunchParams.accumPtr[fbOfs];
    vec4 accum_color = vec4((illum + float(optixLaunchParams.frameID) * vec3(prev_color)) / float(optixLaunchParams.frameID + 1), 1.0f);
    optixLaunchParams.accumPtr[fbOfs] = accum_color;
    optixLaunchParams.fbPtr[fbOfs] = vec4(
        linear_to_srgb(accum_color.x),
        linear_to_srgb(accum_color.y),
        linear_to_srgb(accum_color.z),
        1.0f
    );
}

