#include "hip/hip_runtime.h"
#include "path_tracer.h"
#include "disney_bsdf.h"
#include "lights.h"
#include "launch_params.h"
#include "types.h"
#include <optix_device.h>
#include <owl/common/math/random.h>

typedef owl::common::LCG<4> Random;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct RayPayload {
    uint32_t entityID;
    float2 uv;
    float tHit;
    float3 normal;
    float3 gnormal;
    // float pad;
};

inline __device__
vec2 toSpherical(vec3 dir) {
    dir = normalize(dir);
    float u = atan(dir.z, dir.x) / (2.0 * 3.1415926535897932384626433832795) + .5;
    float v = asin(dir.y) / 3.1415926535897932384626433832795 + .5;
    return vec2(u, (1.0 - v));
}

inline __device__
float3 missColor(const owl::Ray &ray)
{
    // return make_float3(.5f);

    auto pixelID = owl::getLaunchIndex();

    float3 rayDir = normalize(ray.direction);
    if (optixLaunchParams.environmentMapID != -1) 
    {
        vec2 tc = toSpherical(vec3(rayDir.x, -rayDir.z, rayDir.y));
        hipTextureObject_t tex = optixLaunchParams.textureObjects[optixLaunchParams.environmentMapID];
        if (!tex) return make_float3(1.f, 0.f, 1.f);

        float4 texColor = tex2D<float4>(tex, tc.x,tc.y);
        return make_float3(texColor);
    }

    float t = 0.5f*(rayDir.z + 1.0f);
    float3 c = (1.0f - t) * make_float3(1.0f, 1.0f, 1.0f) + t * make_float3(0.5f, 0.7f, 1.0f);
    return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
    RayPayload &payload = get_payload<RayPayload>();
    payload.tHit = -1.f;
    payload.entityID = -1;
    owl::Ray ray;
    ray.direction = optixGetWorldRayDirection();
    payload.normal = missColor(ray) * optixLaunchParams.domeLightIntensity;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    
    const float2 bc    = optixGetTriangleBarycentrics();
    const int instID   = optixGetInstanceIndex();
    const int primID   = optixGetPrimitiveIndex();
    const int entityID = optixLaunchParams.instanceToEntityMap[instID];
    const ivec3 index  = self.index[primID];
    
    // compute position: (actually not needed. implicit via tMax )
    // vec3 V;
    // {
    //     const vec3 &A      = self.vertex[index.x];
    //     const vec3 &B      = self.vertex[index.y];
    //     const vec3 &C      = self.vertex[index.z];
    //     V = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    // }

    // compute normal:
    float3 N, GN;

    const float3 &A      = (float3&) self.vertex[index.x];
    const float3 &B      = (float3&) self.vertex[index.y];
    const float3 &C      = (float3&) self.vertex[index.z];
    GN = normalize(cross(B-A,C-A));
    
    if (self.normals) {
        const float3 &A = (float3&) self.normals[index.x];
        const float3 &B = (float3&) self.normals[index.y];
        const float3 &C = (float3&) self.normals[index.z];
        N = normalize(A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y);
    } else {
        N = GN;
    }

    GN = normalize(optixTransformNormalFromObjectToWorldSpace(GN));
    N = normalize(optixTransformNormalFromObjectToWorldSpace(N));
    // normalize(transpose(mat3(gl_WorldToObjectNV)) * payload.m_n);
    // N  = normalize(transpose(mat3(gl_WorldToObjectNV)) * payload.m_n);

    // compute uv:
    float2 UV;
    if (self.texcoords) {
        const float2 &A = (float2&) self.texcoords[index.x];
        const float2 &B = (float2&) self.texcoords[index.y];
        const float2 &C = (float2&) self.texcoords[index.z];
        UV = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    } else {
        UV = bc;
    }

    // store data in payload
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.entityID = entityID;
    prd.uv = UV;
    prd.tHit = optixGetRayTmax();
    prd.normal = N;
    prd.gnormal = GN;
}

inline __device__
bool loadCamera(EntityStruct &cameraEntity, CameraStruct &camera, TransformStruct &transform)
{
    cameraEntity = optixLaunchParams.cameraEntity;
    if (!cameraEntity.initialized) return false;
    if ((cameraEntity.transform_id < 0) || (cameraEntity.transform_id >= MAX_TRANSFORMS)) return false;
    if ((cameraEntity.camera_id < 0) || (cameraEntity.camera_id >= MAX_CAMERAS)) return false;
    camera = optixLaunchParams.cameras[cameraEntity.camera_id];
    transform = optixLaunchParams.transforms[cameraEntity.transform_id];
    return true;
}

__device__ 
void loadMaterial(const MaterialStruct &p, float2 uv, DisneyMaterial &mat, float roughnessMinimum) {

    // uint32_t mask = __float_as_int(p.base_color.x);
    // if (IS_TEXTURED_PARAM(mask)) {
    //     const uint32_t tex_id = GET_TEXTURE_ID(mask);
    //     mat.base_color = make_float3(tex2D<float4>(launch_params.textures[tex_id], uv.x, uv.y));
    // } else {
        mat.base_color = make_float3(p.base_color.x, p.base_color.y, p.base_color.z);
    // }

    mat.metallic = /*textured_scalar_param(*/p.metallic/*, uv)*/;
    mat.specular = /*textured_scalar_param(*/p.specular/*, uv)*/;
    mat.roughness = max(/*textured_scalar_param(*/max(p.roughness, MIN_ROUGHNESS)/*, uv)*/, roughnessMinimum);
    mat.specular_tint = /*textured_scalar_param(*/p.specular_tint/*, uv)*/;
    mat.anisotropy = /*textured_scalar_param(*/p.anisotropic/*, uv)*/;
    mat.sheen = /*textured_scalar_param(*/p.sheen/*, uv)*/;
    mat.sheen_tint = /*textured_scalar_param(*/p.sheen_tint/*, uv)*/;
    mat.clearcoat = /*textured_scalar_param(*/p.clearcoat/*, uv)*/;
    mat.clearcoat_gloss = /*textured_scalar_param(*/1.0 - max(p.clearcoat_roughness, roughnessMinimum)/*, uv)*/;
    mat.ior = /*textured_scalar_param(*/p.ior/*, uv)*/;
    mat.specular_transmission = /*textured_scalar_param(*/p.transmission/*, uv)*/;
    mat.flatness = p.subsurface;
}

inline __device__
owl::Ray generateRay(const CameraStruct &camera, const TransformStruct &transform, ivec2 pixelID, ivec2 frameSize, LCGRand &rng)
{
    /* Generate camera rays */    
    mat4 camWorldToLocal = transform.worldToLocal;
    mat4 projinv = camera.projinv;//glm::inverse(glm::perspective(.785398, 1.0, .1, 1000));//camera.projinv;
    mat4 viewinv = camera.viewinv * camWorldToLocal;
    vec2 aa = vec2(lcg_randomf(rng),lcg_randomf(rng)) - vec2(.5f,.5f);
    vec2 inUV = (vec2(pixelID.x, pixelID.y) + aa) / vec2(optixLaunchParams.frameSize);
    vec3 right = normalize(glm::vec3(viewinv[0]));
    vec3 up = normalize(glm::vec3(viewinv[1]));
    
    float cameraLensRadius = camera.apertureDiameter;

    vec3 p(0.f);
    if (cameraLensRadius > 0.0) {
        do {
            p = 2.0f*vec3(lcg_randomf(rng),lcg_randomf(rng),0.f) - vec3(1.f,1.f,0.f);
        } while (dot(p,p) >= 1.0f);
    }

    vec3 rd = cameraLensRadius * p;
    vec3 lens_offset = (right * rd.x) / float(frameSize.x) + (up * rd.y) / float(frameSize.y);

    vec3 origin = vec3(viewinv * vec4(0.f,0.f,0.f,1.f)) + lens_offset;
    vec2 dir = inUV * 2.f - 1.f; dir.y *= -1.f;
    vec4 t = (projinv * vec4(dir.x, dir.y, -1.f, 1.f));
    vec3 target = vec3(t) / float(t.w);
    vec3 direction = normalize(vec3(viewinv * vec4(target, 0.f))) * camera.focalDistance;
    direction = normalize(direction - lens_offset);

    owl::Ray ray;
    ray.tmin = .001f;
    ray.tmax = 1e20f;//10000.0f;
    ray.origin = owl::vec3f(origin.x, origin.y, origin.z) ;
    ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    ray.direction = normalize(owl::vec3f(direction.x, direction.y, direction.z));
    
    return ray;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    auto pixelID = ivec2(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    auto fbOfs = pixelID.x+optixLaunchParams.frameSize.x* ((optixLaunchParams.frameSize.y - 1) -  pixelID.y);
    LCGRand rng = get_rng(optixLaunchParams.frameID);

    EntityStruct    camera_entity;
    TransformStruct camera_transform;
    CameraStruct    camera;
    if (!loadCamera(camera_entity, camera, camera_transform)) {
        optixLaunchParams.frameBuffer[fbOfs] = vec4(lcg_randomf(rng), lcg_randomf(rng), lcg_randomf(rng), 1.f);
        return;
    }

    float3 accum_illum = make_float3(0.f);
    float3 primaryAlbedo = make_float3(0.f);
    float3 primaryNormal = make_float3(0.f);
    uint32_t lastSampledLightID = -1;
    #define SPP 1
    for (uint32_t rid = 0; rid < SPP; ++rid) {

        owl::Ray ray = generateRay(camera, camera_transform, pixelID, optixLaunchParams.frameSize, rng);

        DisneyMaterial mat;
        int bounce = 0;
        float3 illum = make_float3(0.f);
        float3 path_throughput = make_float3(1.f);
        uint16_t ray_count = 0;
        float roughnessMinimum = 0.f;
        RayPayload payload;
        owl::traceRay(  /*accel to trace against*/ optixLaunchParams.world,
                        /*the ray to trace*/ ray,
                        /*prd*/ payload);

        do {
            // if ray misses, interpret normal as "miss color" assigned by miss program
            if (payload.tHit <= 0.f) {
                illum = illum + path_throughput * payload.normal;
                break;
            }

            EntityStruct entity = optixLaunchParams.entities[payload.entityID];
            MaterialStruct entityMaterial;
            LightStruct entityLight;
            if (entity.material_id >= 0 && entity.material_id < MAX_MATERIALS) {
                entityMaterial = optixLaunchParams.materials[entity.material_id];
            }
            TransformStruct entityTransform = optixLaunchParams.transforms[entity.transform_id];
            loadMaterial(entityMaterial, payload.uv, mat, roughnessMinimum);

            const float3 w_o = -ray.direction;
            const float3 hit_p = ray.origin + payload.tHit * ray.direction;
            float3 v_x, v_y;
            float3 v_z = payload.normal;
            float3 v_gz = payload.gnormal;
            if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
                // prevents differences from geometric and shading normal from creating black artifacts
                v_z = reflect(-v_z, v_gz); 
            }
            if (mat.specular_transmission == 0.f && dot(w_o, v_z) < 0.f) {
                v_z = -v_z;
            }
            ortho_basis(v_x, v_y, v_z);
            
            // illum = illum + path_throughput * 
            // sample_direct_light(mat, hit_p, v_z, v_x, v_y, w_o,
                //     optixLaunchParams.lights, 
                //     optixLaunchParams.entities, 
                //     optixLaunchParams.transforms, 
                //     optixLaunchParams.meshes, 
                //     optixLaunchParams.lightEntities,
                //     optixLaunchParams.numLightEntities, 
            //     ray_count, rng, 
            //     sampledSpecularLight, sampledLightID);
            
            uint32_t sampledLightID = -1;
            bool sampledSpecularLight = false;
            int numLights = optixLaunchParams.numLightEntities;
            float3 lightEmission = make_float3(0.f);
            float3 irradiance = make_float3(0.f);
            float light_pdf = 0.f;
            do {
                if (numLights == 0) break;
                
                uint32_t random_id = uint32_t(min(lcg_randomf(rng) * numLights, float(numLights - 1)));
                random_id = min(random_id, numLights - 1);
                sampledLightID = optixLaunchParams.lightEntities[random_id];
                EntityStruct light_entity = optixLaunchParams.entities[sampledLightID];
                
                // shouldn't happen, but just in case...
                if ((light_entity.light_id < 0) || (light_entity.light_id > MAX_LIGHTS)) break;
                if ((light_entity.transform_id < 0) || (light_entity.transform_id > MAX_TRANSFORMS)) break;
            
                
                LightStruct light = optixLaunchParams.lights[light_entity.light_id];
                TransformStruct transform = optixLaunchParams.transforms[light_entity.transform_id];
                MeshStruct mesh;
                bool is_area_light = false;
                if ((light_entity.mesh_id >= 0) && (light_entity.mesh_id < MAX_MESHES)) {
                    mesh = optixLaunchParams.meshes[light_entity.mesh_id];
                    is_area_light = true;
                };
            
                lightEmission = make_float3(light.r, light.g, light.b) * light.intensity;
            
                const uint32_t occlusion_flags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
                    // | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT;
                    // | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT;
            
                if (!is_area_light) break;

                uint32_t random_tri_id = uint32_t(min(lcg_randomf(rng) * mesh.numTris, float(mesh.numTris - 1)));
                ivec3* triIndices = optixLaunchParams.indexLists[light_entity.mesh_id]; 
                ivec3 triIndex = triIndices[random_tri_id];   
                
                // Sample the light to compute an incident light ray to this point
                {    
                    glm::mat4 tfm =  transform.localToWorld;//glm::translate(glm::mat4(1.0f), transform.translation) * glm::toMat4(transform.rotation);
                    glm::mat4 tfmInv = transform.worldToLocal;//glm::inverse(tfm);
                    
                    vec3 dir; 
                    vec3 pos = vec3(hit_p.x, hit_p.y, hit_p.z);
                    vec3 v1 = transform.localToWorld * optixLaunchParams.vertexLists[light_entity.mesh_id][triIndex.x];
                    vec3 v2 = transform.localToWorld * optixLaunchParams.vertexLists[light_entity.mesh_id][triIndex.y];
                    vec3 v3 = transform.localToWorld * optixLaunchParams.vertexLists[light_entity.mesh_id][triIndex.z];
                    vec3 N = normalize(cross( normalize(v2 - v1), normalize(v3 - v1)));
                    sampleTriangle(pos, N, v1, v2, v3, lcg_randomf(rng), lcg_randomf(rng), dir, light_pdf);
                    vec3 normal = glm::vec3(v_z.x, v_z.y, v_z.z);
                    float dotNWi = abs(dot(dir, normal));         
        
                    if ((light_pdf > EPSILON) && (dotNWi > EPSILON)) {
                        float3 light_dir = make_float3(dir.x, dir.y, dir.z);
                        light_dir = normalize(light_dir);
                        float bsdf_pdf = disney_pdf(mat, v_z, w_o, light_dir, v_x, v_y);
                        if (bsdf_pdf > EPSILON) {
                            RayPayload payload;
                            payload.entityID = -1;
                            owl::Ray ray;
                            ray.tmin = EPSILON * 10.f;
                            ray.tmax = 1e20f;
                            ray.origin = hit_p;
                            ray.direction = light_dir;
                            owl::traceRay( optixLaunchParams.world, ray, payload, occlusion_flags);
                            bool visible = ((payload.entityID == sampledLightID) || (payload.entityID == -1));
                            if (visible) {
                                float w = power_heuristic(1.f, light_pdf, 1.f, bsdf_pdf);
                                float3 bsdf = disney_brdf(mat, v_z, w_o, light_dir, v_x, v_y, optixLaunchParams.GGX_E_LOOKUP, optixLaunchParams.GGX_E_AVG_LOOKUP);
                                float3 Li = lightEmission * w / light_pdf;
                                irradiance = (bsdf * Li * fabs(dotNWi));
                            }
                        }
                    }
                }
            } while (false);

            
            float3 w_i;
            float pdf;
            bool sampledSpecular;
            float3 bsdf = sample_disney_brdf(mat, v_z, w_o, v_x, v_y, rng, w_i, pdf, sampledSpecular, optixLaunchParams.GGX_E_LOOKUP, optixLaunchParams.GGX_E_AVG_LOOKUP);
            if (pdf < EPSILON || all_zero(bsdf)) {
                break;
            }

            // vec3 offset = payload.normal * .001f;
            ray.origin = hit_p;// + make_float3(offset.x, offset.y, offset.z);
            ray.direction = w_i;
            owl::traceRay(optixLaunchParams.world, ray, payload);

            // Sample the BRDF to compute a light sample as well
            {
                const uint32_t occlusion_flags = OPTIX_RAY_FLAG_DISABLE_ANYHIT;
                float3 w_i;
                float bsdf_pdf;
                float3 bsdf = sample_disney_brdf(mat, v_z, w_o, v_x, v_y, rng, w_i, bsdf_pdf, sampledSpecularLight, optixLaunchParams.GGX_E_LOOKUP, optixLaunchParams.GGX_E_AVG_LOOKUP);
                if ((light_pdf > EPSILON) && !all_zero(bsdf) && bsdf_pdf >= EPSILON) {        
                    bool visible = (payload.entityID == sampledLightID);
                    if (visible) {
                        float w = power_heuristic(1.f, bsdf_pdf, 1.f, light_pdf);
                        irradiance = irradiance + bsdf * lightEmission * fabs(dot(w_i, v_z)) * w / ((payload.tHit * payload.tHit) * bsdf_pdf);
                    }
                }
            }

            if (entity.light_id >= 0 && entity.light_id < MAX_LIGHTS) {
                // Don't double count lights, since we're doing NEE
                // Area lights are only visible outside of NEE sampling when hit on first bounce.
                // TODO: shade light sources, adding on emission later.
                // if (entity.light_id != lastSampledLightID) {
                // } 
                if (bounce == 0) {
                    entityLight = optixLaunchParams.lights[entity.light_id];
                    float3 light_emission = make_float3(entityLight.r, entityLight.g, entityLight.b) * entityLight.intensity;
                    float dist = distance(vec3(hit_p.x, hit_p.y, hit_p.z ), vec3(ray.origin.x, ray.origin.y, ray.origin.z));
                    float dotNWi = dot(v_z, ray.direction);
                    illum = illum + (path_throughput * light_emission * fabs(dotNWi)) / max(dist * dist, EPSILON); // need to compute solid angle here...
                    primaryNormal = payload.normal;
                    primaryAlbedo = mat.base_color;
                }
                break;
            }

            illum = illum + path_throughput * irradiance;
            path_throughput = path_throughput * bsdf / pdf;
            lastSampledLightID = sampledLightID;

            if (bounce == 0) {
                primaryNormal = payload.normal;
                primaryAlbedo = mat.base_color;
            }

            if (path_throughput.x < EPSILON && path_throughput.y < EPSILON && path_throughput.z < EPSILON) {
                break;
            }

            // path regularization to reduce fireflies
            if (sampledSpecular || sampledSpecularLight) {
                roughnessMinimum = min((roughnessMinimum + .35f), 1.f);
            }

            ++bounce;            
        } while (bounce < MAX_PATH_DEPTH);
        // clamp out fireflies
        glm::vec3 gillum = vec3(illum.x, illum.y, illum.z);
        gillum = clamp(gillum, vec3(0.f), vec3(500.f));

        // just in case we get inf's or nans, remove them.
        if (glm::any(glm::isnan(gillum))) gillum = vec3(0.f);
        if (glm::any(glm::isinf(gillum))) gillum = vec3(0.f);
        illum = make_float3(gillum.r, gillum.g, gillum.b);

        accum_illum = accum_illum + illum;
    }
    accum_illum = accum_illum / float(SPP);


    // finalColor = vec3(ray.direction.x, ray.direction.y, ray.direction.z);
    /* Write AOVs */
    float4 &prev_color = (float4&) optixLaunchParams.accumPtr[fbOfs];
    float4 accum_color = make_float4((accum_illum + float(optixLaunchParams.frameID) * make_float3(prev_color)) / float(optixLaunchParams.frameID + 1), 1.0f);
    optixLaunchParams.accumPtr[fbOfs] = vec4(
        accum_color.x, 
        accum_color.y, 
        accum_color.z, 
        accum_color.w
    );
    optixLaunchParams.frameBuffer[fbOfs] = vec4(
        linear_to_srgb(accum_color.x),
        linear_to_srgb(accum_color.y),
        linear_to_srgb(accum_color.z),
        1.0f
    );

    vec4 oldAlbedo = optixLaunchParams.albedoBuffer[fbOfs];
    vec4 oldNormal = optixLaunchParams.normalBuffer[fbOfs];
    vec4 newAlbedo = vec4(primaryAlbedo.x, primaryAlbedo.y, primaryAlbedo.z, 1.f);
    vec4 newNormal = normalize(camera_transform.worldToLocal * vec4(primaryNormal.x, primaryNormal.y, primaryNormal.z, 0.f));
    vec4 accumAlbedo = (newAlbedo + float(optixLaunchParams.frameID) * oldAlbedo) / float(optixLaunchParams.frameID + 1);
    vec4 accumNormal = (newNormal + float(optixLaunchParams.frameID) * oldNormal) / float(optixLaunchParams.frameID + 1);
    optixLaunchParams.albedoBuffer[fbOfs] = accumAlbedo;
    optixLaunchParams.normalBuffer[fbOfs] = accumNormal;
}

