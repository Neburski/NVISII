#include "hip/hip_runtime.h"
#include "path_tracer.h"
#include "disney_bsdf.h"
#include "lights.h"
#include "launch_params.h"
#include <optix_device.h>
#include <owl/common/math/random.h>

typedef owl::common::LCG<4> Random;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct RayPayload {
    vec2 uv;
    float tHit;
    uint32_t entityId;
    vec3 normal;
    // float pad;
};

inline __device__
vec3 missColor(const owl::Ray &ray)
{
  auto pixelID = owl::getLaunchIndex();

  vec3 rayDir = glm::normalize(glm::vec3(ray.direction.x, ray.direction.y, ray.direction.z));
  float t = 0.5f*(rayDir.y + 1.0f);
  vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    
    const float2 bc    = optixGetTriangleBarycentrics();
    const int instID   = optixGetInstanceIndex();
    const int primID   = optixGetPrimitiveIndex();
    const int entityID = optixLaunchParams.instanceToEntityMap[instID];
    const ivec3 index  = self.index[primID];
    
    // compute position: (actually not needed. implicit via tMax )
    // vec3 V;
    // {
    //     const vec3 &A      = self.vertex[index.x];
    //     const vec3 &B      = self.vertex[index.y];
    //     const vec3 &C      = self.vertex[index.z];
    //     V = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    // }

    // compute normal:
    vec3 N;
    if (self.normals) {
        const vec3 &A = self.normals[index.x];
        const vec3 &B = self.normals[index.y];
        const vec3 &C = self.normals[index.z];
        N = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    } else {
        const vec3 &A      = self.vertex[index.x];
        const vec3 &B      = self.vertex[index.y];
        const vec3 &C      = self.vertex[index.z];
        N = normalize(cross(B-A,C-A));
    }

    // compute uv:
    vec2 UV;
    if (self.texcoords) {
        const vec2 A = self.texcoords[index.x];
        const vec2 B = self.texcoords[index.y];
        const vec2 C = self.texcoords[index.z];
        UV = A * (1.f - (bc.x + bc.y)) + B * bc.x + C * bc.y;
    } else {
        UV = vec2(bc.x, bc.y);
    }

    // store data in payload
    RayPayload &prd = owl::getPRD<RayPayload>();
    prd.uv = UV;
    prd.tHit = optixGetRayTmax();
    prd.normal = N;
}

inline __device__
bool loadCamera(EntityStruct &cameraEntity, CameraStruct &camera, TransformStruct &transform)
{
    cameraEntity = optixLaunchParams.cameraEntity;
    if (!cameraEntity.initialized) return false;
    if ((cameraEntity.transform_id < 0) || (cameraEntity.transform_id >= MAX_TRANSFORMS)) return false;
    if ((cameraEntity.camera_id < 0) || (cameraEntity.camera_id >= MAX_CAMERAS)) return false;
    camera = optixLaunchParams.cameras[cameraEntity.camera_id];
    transform = optixLaunchParams.transforms[cameraEntity.transform_id];
    return true;
}

inline __device__
owl::Ray generateRay(const CameraStruct &camera, const TransformStruct &transform, ivec2 pixelID, ivec2 frameSize)
{
    /* Generate camera rays */    
    mat4 camWorldToLocal = transform.worldToLocal;
    mat4 projinv = camera.projinv;//glm::inverse(glm::perspective(.785398, 1.0, .1, 1000));//camera.projinv;
    mat4 viewinv = camera.viewinv * camWorldToLocal;
    vec2 inUV = vec2(pixelID.x, pixelID.y) / vec2(optixLaunchParams.frameSize);
    // if (optixLaunchParams.zoom > 0.f) {
    //     inUV /= optixLaunchParams.zoom;
    //     inUV += (.5f - (.5f / optixLaunchParams.zoom));
    // }

    vec3 origin = vec3(viewinv * vec4(0.f,0.f,0.f,1.f));

    vec2 dir = inUV * 2.f - 1.f; dir.y *= -1.f;
    vec4 t = (projinv * vec4(dir.x, dir.y, -1.f, 1.f));
    vec3 target = vec3(t) / float(t.w);
    vec3 direction = vec3(viewinv * vec4(target, 0.f));
    direction = normalize(direction);

    owl::Ray ray;
    ray.tmin = .0f;
    ray.tmax = 1e38f;//10000.0f;
    ray.origin = owl::vec3f(origin.x, origin.y, origin.z);
    ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    // ray.direction = owl::vec3f(0.0, 1.0, 0.0); // testing...
    // if ((pixelID.x == 0) && (pixelID.y == 0)) {
    //     // printf("dir: %f %f %f\n", ray.direction.x, ray.direction.y, ray.direction.z);
    //     printf("viewinv: %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f\n", 
    //         viewinv[0][0], viewinv[0][1], viewinv[0][2], viewinv[0][3],
    //         viewinv[1][0], viewinv[1][1], viewinv[1][2], viewinv[1][3],
    //         viewinv[2][0], viewinv[2][1], viewinv[2][2], viewinv[2][3],
    //         viewinv[3][0], viewinv[3][1], viewinv[3][2], viewinv[3][3]
    //     );
    // }

    ray.direction = normalize(owl::vec3f(direction.x, direction.y, direction.z));
    // ray.direction = normalize(owl::vec3f(target.x, target.y, target.z));

    // vec3 lookFrom = origin;//(-4.f,-3.f,-2.f);
    // vec3 lookAt(0.f,0.f,0.f);
    // vec3 lookUp(0.f,0.f,1.f);
    // float cosFovy = 0.66f;
    // vec3 camera_pos = lookFrom;
    // vec3 camera_d00
    //   = normalize(lookAt-lookFrom);
    // float aspect = frameSize.x / float(frameSize.y);
    // vec3 camera_ddu
    //   = cosFovy * aspect * normalize(cross(camera_d00,lookUp));
    // vec3 camera_ddv
    //   = cosFovy * normalize(cross(camera_ddu,camera_d00));
    // camera_d00 -= 0.5f * camera_ddu;
    // camera_d00 -= 0.5f * camera_ddv;

    // direction 
    // = normalize(camera_d00
    //             + inUV.x * camera_ddu
    //             + inUV.y * camera_ddv);
    // ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    return ray;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    auto pixelID = ivec2(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    auto fbOfs = pixelID.x+optixLaunchParams.frameSize.x* ((optixLaunchParams.frameSize.y - 1) -  pixelID.y);
    LCGRand rng = get_rng(optixLaunchParams.frameID);

    EntityStruct    camera_entity;
    TransformStruct camera_transform;
    CameraStruct    camera;
    if (!loadCamera(camera_entity, camera, camera_transform)) {
        optixLaunchParams.fbPtr[fbOfs] = vec4(lcg_randomf(rng), lcg_randomf(rng), lcg_randomf(rng), 1.f);
        return;
    }

    owl::Ray ray = generateRay(camera, camera_transform, pixelID, optixLaunchParams.frameSize);

    vec3 finalColor = vec3(0.f);

    /* Intersect mesh */
    RayPayload tprd;
    owl::traceRay(  /*accel to trace against*/ optixLaunchParams.world,
                    /*the ray to trace*/ ray,
                    /*prd*/ tprd);  
    
    if (tprd.tHit > 0.f) {
        finalColor = vec3(tprd.normal.x, tprd.normal.y, tprd.normal.z);
    }
    // finalColor = vec3(ray.direction.x, ray.direction.y, ray.direction.z);
    /* Write AOVs */
    optixLaunchParams.fbPtr[fbOfs] = vec4(finalColor.r, finalColor.g, finalColor.b, 1.f);
}

