#include "hip/hip_runtime.h"
#include "deviceCode.h"
#include "launchParams.h"
#include <optix_device.h>
#include <owl/common/math/random.h>
typedef owl::common::LCG<4> Random;

extern "C" __constant__ LaunchParams optixLaunchParams;

struct TriMeshPayload {
    float r = -1.f, g = -1.f, b = -1.f, tmax = -1.f;
};

inline __device__
vec3 missColor(const owl::Ray &ray)
{
  auto pixelID = owl::getLaunchIndex();

  vec3 rayDir = glm::normalize(glm::vec3(ray.direction.x, ray.direction.y, ray.direction.z));
  float t = 0.5f*(rayDir.y + 1.0f);
  vec3 c = (1.0f - t) * vec3(1.0f, 1.0f, 1.0f) + t * vec3(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
    TriMeshPayload &prd = owl::getPRD<TriMeshPayload>();

    const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
    float2 bc = optixGetTriangleBarycentrics();

    // compute normal:
    const int   primID = optixGetPrimitiveIndex();
    const ivec3 index  = self.index[primID];
    const vec3 &A     = self.vertex[index.x];
    const vec3 &B     = self.vertex[index.y];
    const vec3 &C     = self.vertex[index.z];
    const vec3 &ACol = vec3(1.0, 0.0, 0.0); ///(self.colors == nullptr) ? vec3(optixLaunchParams.tri_mesh_color) : self.colors[index.x];
    const vec3 &BCol = vec3(1.0, 0.0, 0.0); ///(self.colors == nullptr) ? vec3(optixLaunchParams.tri_mesh_color) : self.colors[index.y];
    const vec3 &CCol = vec3(1.0, 0.0, 0.0); ///(self.colors == nullptr) ? vec3(optixLaunchParams.tri_mesh_color) : self.colors[index.z];
    const vec3 Ng     = normalize(cross(B-A,C-A));

    auto rayDir = optixGetWorldRayDirection();
    vec3 dir = vec3(rayDir.x, rayDir.y, rayDir.z);

    vec3 vcol = ACol * (1.f - (bc.x + bc.y)) + BCol * bc.x + CCol * bc.y;

    vec3 color = (.2f + .8f*fabs(dot(dir,Ng)))*vcol;
    prd.r = color.x;
    prd.g = color.y;
    prd.b = color.z;
    prd.tmax = optixGetRayTmax();
}

inline __device__
bool loadCamera(EntityStruct &cameraEntity, CameraStruct &camera, TransformStruct &transform)
{
    cameraEntity = optixLaunchParams.cameraEntity;
    if (!cameraEntity.initialized) return false;
    if ((cameraEntity.transform_id < 0) || (cameraEntity.transform_id >= MAX_TRANSFORMS)) return false;
    if ((cameraEntity.camera_id < 0) || (cameraEntity.camera_id >= MAX_CAMERAS)) return false;
    camera = optixLaunchParams.cameras[cameraEntity.camera_id];
    transform = optixLaunchParams.transforms[cameraEntity.transform_id];
    return true;
}

inline __device__
owl::Ray generateRay(const CameraStruct &camera, const TransformStruct &transform, ivec2 pixelID, ivec2 frameSize)
{
    /* Generate camera rays */    
    mat4 camWorldToLocal = transform.worldToLocal;
    mat4 projinv = camera.projinv;
    mat4 viewinv = camera.viewinv * camWorldToLocal;
    vec2 inUV = vec2(pixelID.x, pixelID.y) / vec2(optixLaunchParams.frameSize);
    // if (optixLaunchParams.zoom > 0.f) {
    //     inUV /= optixLaunchParams.zoom;
    //     inUV += (.5f - (.5f / optixLaunchParams.zoom));
    // }
    vec2 dir = inUV * 2.f - 1.f; dir.y *= -1.f;
    vec4 t = (projinv * vec4(dir.x, dir.y, 1.f, 1.f));
    vec3 target = vec3(t) / float(t.w);
    vec3 origin = vec3(viewinv * vec4(0.f,0.f,0.f,1.f));
    vec3 direction = vec3(viewinv * vec4(target, 0.f));
    direction = normalize(direction);

    owl::Ray ray;
    ray.tmin = .0f;
    ray.tmax = 1e38f;//10000.0f;
    ray.origin = owl::vec3f(origin.x, origin.y, origin.z);
    ray.direction = owl::vec3f(direction.x, direction.y, direction.z);
    if ((pixelID.x == 0) && (pixelID.y == 0)) {
        // printf("dir: %f %f %f\n", ray.direction.x, ray.direction.y, ray.direction.z);
        printf("viewinv: %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f %f\n", 
            viewinv[0][0], viewinv[0][1], viewinv[0][2], viewinv[0][3],
            viewinv[1][0], viewinv[1][1], viewinv[1][2], viewinv[1][3],
            viewinv[2][0], viewinv[2][1], viewinv[2][2], viewinv[2][3],
            viewinv[3][0], viewinv[3][1], viewinv[3][2], viewinv[3][3]
        );
    }
    return ray;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
    auto pixelID = ivec2(owl::getLaunchIndex()[0], owl::getLaunchIndex()[1]);
    auto fbOfs = pixelID.x+optixLaunchParams.frameSize.x* ((optixLaunchParams.frameSize.y - 1) -  pixelID.y);
    Random random; random.init(pixelID.x/* + offset*/, pixelID.y/* + offset*/);

    EntityStruct    camera_entity;
    TransformStruct camera_transform;
    CameraStruct    camera;
    if (!loadCamera(camera_entity, camera, camera_transform)) {
        optixLaunchParams.fbPtr[fbOfs] = vec4(random(), random(), random(), 1.f);
        return;
    }

    owl::Ray ray = generateRay(camera, camera_transform, pixelID, optixLaunchParams.frameSize);

    /* Write AOVs */
    optixLaunchParams.fbPtr[fbOfs] = vec4(ray.direction.x, ray.direction.y, ray.direction.z, 1.f);
}

